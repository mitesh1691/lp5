
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

using namespace std;

__global__ void add(int* A, int* B, int* C, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        C[tid] = A[tid] + B[tid];
    }
}

// Original initialize function
/*
void initialize(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        vector[i] = rand() % 10;
    }
}
*/

// New function to initialize vectors with random elements
void initializeRandom(int* vector, int size) {
    srand(time(0));
    for (int i = 0; i < size; i++) {
        vector[i] = rand() % 100; // Generating random numbers between 0 to 99
    }
}

void print(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        cout << vector[i] << " ";
    }
    cout << endl;
}

int main() {
    int N;
    cout << "Enter the size of the vectors: ";
    cin >> N;

    int* A, * B, * C;

    int vectorSize = N;
    size_t vectorBytes = vectorSize * sizeof(int);

    A = new int[vectorSize];
    B = new int[vectorSize];
    C = new int[vectorSize];

    // Use the new function to initialize vectors with random elements
    initializeRandom(A, vectorSize);
    initializeRandom(B, vectorSize);

    cout << "Vector A: ";
    print(A, N);
    cout << "Vector B: ";
    print(B, N);

    int* X, * Y, * Z;
    hipMalloc(&X, vectorBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);

    hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);

    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);

    cout << "Addition: ";
    print(C, N);

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}



// how to run: 
// open terminal - 

// run this: 
// nvcc code.cu -o code

// then this: 
// ./code

// NOTE - save the .cu file in the home directory only (file manager open karne par jo open hota hai wahi pe save karna .cu file kahi bhi mat save kro)