

#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

using namespace std;

__global__ void add(int* A, int* B, int* C, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        C[tid] = A[tid] + B[tid];
    }
}

void initializeRandom(int* vector, int size, int seed) {
    srand(seed);
    for (int i = 0; i < size; i++) {
        vector[i] = rand() % 100; // Generating random numbers between 0 to 99
    }
}

void print(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        cout << vector[i] << " ";
    }
    cout << endl;
}

int main() {
    int N;
    cout << "Enter the size of the vectors: ";
    cin >> N;

    int* A, * B, * C;

    int vectorSize = N;
    size_t vectorBytes = vectorSize * sizeof(int);

    A = new int[vectorSize];
    B = new int[vectorSize];
    C = new int[vectorSize];

    // Use the new function to initialize vectors with different random elements
    initializeRandom(A, vectorSize, 1234); // Use different seed for vector A
    initializeRandom(B, vectorSize, 5678); // Use different seed for vector B

    cout << "Vector A: ";
    print(A, N);
    cout << "Vector B: ";
    print(B, N);

    int* X, * Y, * Z;
    hipMalloc(&X, vectorBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);

    hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);

    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);

    cout << "Addition: ";
    print(C, N);

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}


// how to run: 
// open terminal - 

// run this: 
// nvcc code.cu -o code

// then this: 
// ./code

// NOTE - save the .cu file in the home directory only (file manager open karne par jo open hota hai wahi pe save karna .cu file kahi bhi mat save kro)
