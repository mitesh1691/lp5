
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <ctime>

using namespace std;

__global__ void add(int* A, int* B, int* C, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        C[tid] = A[tid] + B[tid];
    }
}

__global__ void multiply(int* A, int* B, int* C, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        C[tid] = A[tid] * B[tid];
    }
}

// Original initialize function
/*
void initialize(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        vector[i] = rand() % 10;
    }
}
*/

// New function to initialize vectors with random elements
void initializeRandom(int* vector, int size) {
    srand(time(0));
    for (int i = 0; i < size; i++) {
        vector[i] = rand() % 100; // Generating random numbers between 0 to 99
    }
}

void print(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        cout << vector[i] << " ";
    }
    cout << endl;
}

int main() {
    int N;
    cout << "Enter the size of the vectors: ";
    cin >> N;

    int* A, * B, * C, * D;

    int vectorSize = N;
    size_t vectorBytes = vectorSize * sizeof(int);

    A = new int[vectorSize];
    B = new int[vectorSize];
    C = new int[vectorSize];
    D = new int[vectorSize];

    // Use the new function to initialize vectors with random elements
    initializeRandom(A, vectorSize);
    initializeRandom(B, vectorSize);

    cout << "Vector A: ";
    print(A, N);
    cout << "Vector B: ";
    print(B, N);

    int* X, * Y, * Z;
    hipMalloc(&X, vectorBytes);
    hipMalloc(&Y, vectorBytes);
    hipMalloc(&Z, vectorBytes);

    hipMemcpy(X, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(Y, B, vectorBytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Perform addition on GPU
    add<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);

    // Perform multiplication on GPU
    multiply<<<blocksPerGrid, threadsPerBlock>>>(X, Y, Z, N);

    hipMemcpy(C, Z, vectorBytes, hipMemcpyDeviceToHost);

    cout << "Addition: ";
    print(C, N);

    // Clean up
    delete[] A;
    delete[] B;
    delete[] C;
    delete[] D;

    hipFree(X);
    hipFree(Y);
    hipFree(Z);

    return 0;
}



// how to run: 
// open terminal - 

// run this: 
// nvcc code.cu -o code

// then this: 
// ./code

// NOTE - save the .cu file in the home directory only (file manager open karne par jo open hota hai wahi pe save karna .cu file kahi bhi mat save kro)